#include "hip/hip_runtime.h"
#include "myCudaForQt.h"
#include <chrono>
/**/
static __global__ void cuSeparateLR(const int8_t* idata, int8_t* odata, const size_t dataCountTotal)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int targetIdx = idx;
    if (targetIdx%2 == 1) targetIdx += dataCountTotal;
    targetIdx >>= 1;
    odata[targetIdx] = idata[idx];
}
static __global__ void cuSeparateLR(const int16_t* idata, int16_t* odata, const size_t dataCountTotal)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int targetIdx = idx;
    if (targetIdx%2 == 1) targetIdx += dataCountTotal;
    targetIdx >>= 1;
    odata[targetIdx] = idata[idx];
}

static __global__ void cuScatterLR(const int8_t* idata, int8_t* odata, const size_t dataCountTotal)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int targetIdx = idx;
    targetIdx <<= 1;
    if (targetIdx >= dataCountTotal) {
        targetIdx -= dataCountTotal;
        ++targetIdx;
    }
    odata[targetIdx] = idata[idx];
}
static __global__ void cuScatterLR(const int16_t* idata, int16_t* odata, const size_t dataCountTotal)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int targetIdx = idx;
    targetIdx <<= 1;
    if (targetIdx >= dataCountTotal) {
        targetIdx -= dataCountTotal;
        ++targetIdx;
    }
    odata[targetIdx] = idata[idx];
}

static __global__ void cuCopyPCM2R(const int8_t* idata, hipfftReal* odata, const size_t dataCountPerChannel)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    odata[idx] = (hipfftReal)idata[idx]/dataCountPerChannel;
}
static __global__ void cuCopyPCM2R(const int16_t* idata, hipfftReal* odata, const size_t dataCountPerChannel)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    odata[idx] = (hipfftReal)idata[idx]/dataCountPerChannel;
}

static __global__ void cuCopyR2PCM(const hipfftReal* idata, int8_t* odata)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    odata[idx] = (int8_t)std::round(idata[idx]);
}
static __global__ void cuCopyR2PCM(const hipfftReal* idata, int16_t* odata)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    odata[idx] = (int16_t)std::round(idata[idx]);
}

static __global__ void cuMultFilterInplace(hipfftReal* data, const float* filter)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    data[idx] *= filter[idx];
}
static __global__ void cuMultFilterInplace(hipComplex* data, const float* filter)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    data[idx].x *= filter[idx];
    data[idx].y *= filter[idx];
}

static __global__ void cuAddInplace(const int8_t* idata, int8_t* odata)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    odata[idx] += idata[idx];
}
static __global__ void cuAddInplace(const int16_t* idata, int16_t* odata)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    odata[idx] += idata[idx];
}

static __global__ void cuSqrtComplex_8(const hipfftComplex* idata, hipfftReal* odata)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    odata[idx] = log2f(std::hypotf(idata[idx].x, idata[idx].y)*2)/8;
    if (odata[idx] < 0) odata[idx] = 0;
    if (odata[idx] > 1) odata[idx] = 1;
}
static __global__ void cuSqrtComplex_16(const hipfftComplex* idata, hipfftReal* odata)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    odata[idx] = log2f(std::hypotf(idata[idx].x, idata[idx].y)*2)/15;
    if (odata[idx] < 0) odata[idx] = 0;
    if (odata[idx] > 1) odata[idx] = 1;
}

static __global__ void cuInitFloatFilter(float* data, const float initValue)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    data[idx] = initValue;
}
static __global__ void cuMultFloatWeight(float* data, const float weight)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    data[idx] *= weight;
}

static __global__ void cuNormPCM_8(const uint8_t* idata, float* odata) // [0, 1]
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    odata[idx] = (float)idata[idx]/(1ULL<<8);
    if (odata[idx] < 0) odata[idx] = 0;
    if (odata[idx] > 1) odata[idx] = 1;
}
static __global__ void cuNormPCM_16(const int16_t* idata, float* odata) // [-1, 1]
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    odata[idx] = (float)idata[idx]/(1ULL<<15);
    if (odata[idx] < -1) odata[idx] = -1;
    if (odata[idx] > 1) odata[idx] = 1;
}

// static __global__ void cuNormSqrtComplex_8(const hipfftComplex* idata, float* odata) // [0, 1]
// {
//     unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
//     odata[idx] = std::hypotf(idata[idx].x, idata[idx].y)/(1ULL<<8);
//     if (odata[idx] < 0) odata[idx] = 0;
//     if (odata[idx] > 1) odata[idx] = 1;
// }
// static __global__ void cuNormSqrtComplex_16(const hipfftComplex* idata, float* odata) // [0, 1]
// {
//     unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
//     odata[idx] = std::hypotf(idata[idx].x, idata[idx].y)/(1ULL<<15);
//     if (odata[idx] < 0) odata[idx] = 0;
//     if (odata[idx] > 1) odata[idx] = 1;
// }
/*
__host__ void myCUDA::cuFFT(const int8_t* host_pcm, hipfftComplex*& host_freq, const size_t dataCountPerChannel)
{
    // host memory allocation
    if (host_pcm == nullptr) return;
    if (host_freq == nullptr) host_freq = (hipfftComplex*)malloc(sizeof(hipfftComplex)*(dataCountPerChannel/2+1));
    
    // device memory allocation
    int8_t* dev_pcm;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm), sizeof(int8_t)*dataCountPerChannel));
    checkCudaErrors(hipMemcpy(dev_pcm, host_pcm, sizeof(int8_t)*dataCountPerChannel, hipMemcpyHostToDevice));
    hipfftReal* dev_iR;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_iR), sizeof(hipfftReal)*dataCountPerChannel));
    hipfftComplex* dev_oC;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_oC), sizeof(hipfftComplex)*(dataCountPerChannel/2+1)));

    // copy dev_pcm to dev_iR
    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = dataCountPerChannel/threadsPerBlock;
    cuCopyPCM2R<<<blockCount, threadsPerBlock>>>(dev_pcm, dev_iR, dataCountPerChannel); // kernel launch with dataCount blocks

    // plan & FFT
    hipfftHandle plan;
    checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel, HIPFFT_R2C, 1));
    checkCudaErrors(hipfftExecR2C(plan, dev_iR, dev_oC));
    checkCudaErrors(hipfftDestroy(plan));

    // copy transformed result from device to host
    checkCudaErrors(hipMemcpy(host_freq, dev_oC, sizeof(hipfftComplex)*(dataCountPerChannel/2+1), hipMemcpyDeviceToHost));

    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_pcm));
    checkCudaErrors(hipFree(dev_iR));
    checkCudaErrors(hipFree(dev_oC));
}
__host__ void myCUDA::cuFFT(const int16_t* host_pcm, hipfftComplex*& host_freq, const size_t dataCountPerChannel)
{
    // host memory allocation
    if (host_pcm == nullptr) return;
    if (host_freq == nullptr) host_freq = (hipfftComplex*)malloc(sizeof(hipfftComplex)*(dataCountPerChannel/2+1));
    
    // device memory allocation
    int16_t* dev_pcm;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm), sizeof(int16_t)*dataCountPerChannel));
    checkCudaErrors(hipMemcpy(dev_pcm, host_pcm, sizeof(int16_t)*dataCountPerChannel, hipMemcpyHostToDevice));
    hipfftReal* dev_iR;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_iR), sizeof(hipfftReal)*dataCountPerChannel));
    hipfftComplex* dev_oC;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_oC), sizeof(hipfftComplex)*(dataCountPerChannel/2+1)));

    // copy dev_pcm to dev_iR
    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = dataCountPerChannel/threadsPerBlock;
    cuCopyPCM2R<<<blockCount, threadsPerBlock>>>(dev_pcm, dev_iR, dataCountPerChannel); // kernel launch with dataCount blocks

    // plan & FFT
    hipfftHandle plan;
    checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel, HIPFFT_R2C, 1));
    checkCudaErrors(hipfftExecR2C(plan, dev_iR, dev_oC));
    checkCudaErrors(hipfftDestroy(plan));

    // copy transformed result from device to host
    checkCudaErrors(hipMemcpy(host_freq, dev_oC, sizeof(hipfftComplex)*(dataCountPerChannel/2+1), hipMemcpyDeviceToHost));

    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_pcm));
    checkCudaErrors(hipFree(dev_iR));
    checkCudaErrors(hipFree(dev_oC));
}

__host__ void myCUDA::cuFreqFilter(const hipfftComplex* host_freq_i, hipfftComplex*& host_freq_o, const size_t dataCountPerChannel, const float* host_filter)
{
    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = (dataCountPerChannel/2+threadsPerBlock)/threadsPerBlock;

    // host memory allocation
    if (host_filter == nullptr) return;
    if (host_freq_i == nullptr) return;
    if (host_freq_o == nullptr)
        host_freq_o = (hipfftComplex*)malloc(sizeof(hipfftComplex)*(dataCountPerChannel/2+1));

    // device memory allocation
    hipfftComplex* dev_freq;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*(dataCountPerChannel/2+threadsPerBlock)));
    checkCudaErrors(hipMemcpy(dev_freq, host_freq_i, sizeof(hipfftComplex)*(dataCountPerChannel/2+1), hipMemcpyHostToDevice));
    float* dev_filter;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_filter), sizeof(float)*(dataCountPerChannel/2+threadsPerBlock)));
    checkCudaErrors(hipMemcpy(dev_filter, host_filter, sizeof(float)*(dataCountPerChannel/2+1), hipMemcpyHostToDevice));

    // multiply filter
    cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_freq, dev_filter);

    // copy result from device to host
    checkCudaErrors(hipMemcpy(host_freq_o, dev_freq, sizeof(hipfftComplex)*(dataCountPerChannel/2+1), hipMemcpyDeviceToHost));

    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_freq));
    checkCudaErrors(hipFree(dev_filter));
}

__host__ void myCUDA::cuIFFT(const hipfftComplex* host_freq, int8_t*& host_pcm, const size_t dataCountPerChannel)
{
    // host memory allocation
    if (host_freq == nullptr) return;
    if (host_pcm == nullptr)
        host_pcm = (int8_t*)malloc(sizeof(int8_t)*dataCountPerChannel);

    // device memory allocation
    hipfftComplex* dev_iC;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_iC), sizeof(hipfftComplex)*(dataCountPerChannel/2+1)));
    checkCudaErrors(hipMemcpy(dev_iC, host_freq, sizeof(hipfftComplex)*(dataCountPerChannel/2+1), hipMemcpyHostToDevice));
    hipfftReal* dev_oR;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_oR), sizeof(hipfftReal)*dataCountPerChannel));
    int8_t* dev_pcm;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm), sizeof(int8_t)*dataCountPerChannel));

    // plan & FFT
    hipfftHandle plan;
    checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel, HIPFFT_C2R, 1));
    checkCudaErrors(hipfftExecC2R(plan, dev_iC, dev_oR));
    checkCudaErrors(hipfftDestroy(plan));

    // copy transformed result from device to host
    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = dataCountPerChannel/threadsPerBlock;
    cuCopyR2PCM<<<blockCount, threadsPerBlock>>>(dev_oR, dev_pcm);
    checkCudaErrors(hipMemcpy(host_pcm, dev_pcm, sizeof(int8_t)*dataCountPerChannel, hipMemcpyDeviceToHost));

    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_iC));
    checkCudaErrors(hipFree(dev_oR));
    checkCudaErrors(hipFree(dev_pcm));
}
__host__ void myCUDA::cuIFFT(const hipfftComplex* host_freq, int16_t*& host_pcm, const size_t dataCountPerChannel)
{
    // host memory allocation
    if (host_freq == nullptr) return;
    if (host_pcm == nullptr)
        host_pcm = (int16_t*)malloc(sizeof(int16_t)*dataCountPerChannel);

    // device memory allocation
    hipfftComplex* dev_iC;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_iC), sizeof(hipfftComplex)*(dataCountPerChannel/2+1)));
    checkCudaErrors(hipMemcpy(dev_iC, host_freq, sizeof(hipfftComplex)*(dataCountPerChannel/2+1), hipMemcpyHostToDevice));
    hipfftReal* dev_oR;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_oR), sizeof(hipfftReal)*dataCountPerChannel));
    int16_t* dev_pcm;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm), sizeof(int16_t)*dataCountPerChannel));

    // plan & FFT
    hipfftHandle plan;
    checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel, HIPFFT_C2R, 1));
    checkCudaErrors(hipfftExecC2R(plan, dev_iC, dev_oR));
    checkCudaErrors(hipfftDestroy(plan));

    // copy transformed result from device to host
    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = dataCountPerChannel/threadsPerBlock;
    cuCopyR2PCM<<<blockCount, threadsPerBlock>>>(dev_oR, dev_pcm);
    checkCudaErrors(hipMemcpy(host_pcm, dev_pcm, sizeof(int16_t)*dataCountPerChannel, hipMemcpyDeviceToHost));

    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_iC));
    checkCudaErrors(hipFree(dev_oR));
    checkCudaErrors(hipFree(dev_pcm));
}
*/
void myCUDA::printCudaDevice(int argc, char **argv)
{
    printf("Printing CUDA Device... [%s]\n", __FILE__);
    findCudaDevice(argc, (const char **)argv);
}

void myCUDA::afterRecord_8(const int8_t* host_pcm1, const int8_t* host_pcm2, const float* host_hannFilter, void*& host_freq, const size_t dataCountPerChannel, const size_t channelCount)
{
    // [afterRecord]
    // (host pcm) 
    // 두개 연결 -> 채널 분리 -> 채널별 한필터 -> fft 
    // (host freq)

    // host memory allocation
    if (host_hannFilter == nullptr) return;
    if (host_pcm1 == nullptr) return;
    if (host_pcm2 == nullptr) return;
    // if (host_freq == nullptr) host_freq = malloc(sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)*channelCount);
    if (host_freq == nullptr) checkCudaErrors(hipHostMalloc(&host_freq, sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)*channelCount));

    if (channelCount == 1)
    {
        // allocate device memory
        int8_t* dev_pcm;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm), sizeof(int8_t)*dataCountPerChannel*2));
        checkCudaErrors(hipMemcpy(dev_pcm, host_pcm1, sizeof(int8_t)*dataCountPerChannel, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dev_pcm + dataCountPerChannel, host_pcm2, sizeof(int8_t)*dataCountPerChannel, hipMemcpyHostToDevice));
        
        constexpr size_t threadsPerBlock = 32;
        size_t blockCount = dataCountPerChannel*2/threadsPerBlock;

        // PCM to Real
        hipfftReal* dev_real;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_real), sizeof(hipfftReal)*dataCountPerChannel*2));
        cuCopyPCM2R<<<blockCount, threadsPerBlock>>>(dev_pcm, dev_real, dataCountPerChannel*2);

        // Hann filter
        float* dev_hannFilter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_hannFilter), sizeof(float)*dataCountPerChannel*2));
        checkCudaErrors(hipMemcpy(dev_hannFilter, host_hannFilter, sizeof(float)*dataCountPerChannel*2, hipMemcpyHostToDevice));
        cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_real, dev_hannFilter);

        // FFT
        hipfftComplex* dev_freq;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)));
        hipfftHandle plan;
        checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel*2, HIPFFT_R2C, 1));
        checkCudaErrors(hipfftExecR2C(plan, dev_real, dev_freq));
        checkCudaErrors(hipfftDestroy(plan));

        // copy transformed result from device to host
        checkCudaErrors(hipMemcpy(host_freq, dev_freq, sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1), hipMemcpyDeviceToHost));

        // free device(gpu) memory
        checkCudaErrors(hipFree(dev_pcm));
        checkCudaErrors(hipFree(dev_real));
        checkCudaErrors(hipFree(dev_hannFilter));
        checkCudaErrors(hipFree(dev_freq));
    }
    if (channelCount == 2)
    {
        // allocate device memory
        int8_t* dev_pcm_scatter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_scatter), sizeof(int8_t)*dataCountPerChannel*channelCount*2));
        checkCudaErrors(hipMemcpy(dev_pcm_scatter, host_pcm1, sizeof(int8_t)*dataCountPerChannel*channelCount, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dev_pcm_scatter + dataCountPerChannel*channelCount, host_pcm2, sizeof(int8_t)*dataCountPerChannel*channelCount, hipMemcpyHostToDevice));
        int8_t* dev_pcm_separate;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_separate), sizeof(int8_t)*dataCountPerChannel*channelCount*2));

        // separate channel LLLLL...RRRR...
        constexpr size_t threadsPerBlock = 32;
        size_t blockCount = dataCountPerChannel*channelCount*2/threadsPerBlock;
        cuSeparateLR<<<blockCount, threadsPerBlock>>>(dev_pcm_scatter, dev_pcm_separate, dataCountPerChannel*channelCount*2);
        // int8_t* dev_pcm_L = dev_pcm_separate;
        // int8_t* dev_pcm_R = dev_pcm_separate + dataCountPerChannel*2;

        // PCM to Real
        hipfftReal* dev_real;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_real), sizeof(hipfftReal)*dataCountPerChannel*channelCount*2));
        cuCopyPCM2R<<<blockCount, threadsPerBlock>>>(dev_pcm_separate, dev_real, dataCountPerChannel*2);
        hipfftReal* dev_real_L = dev_real;
        hipfftReal* dev_real_R = dev_real + dataCountPerChannel*2;

        // Hann filter each channel
        blockCount = dataCountPerChannel*2/threadsPerBlock;
        float* dev_hannFilter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_hannFilter), sizeof(float)*dataCountPerChannel*2));
        checkCudaErrors(hipMemcpy(dev_hannFilter, host_hannFilter, sizeof(float)*dataCountPerChannel*2, hipMemcpyHostToDevice));
        cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_real_L, dev_hannFilter);
        cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_real_R, dev_hannFilter);

        // FFT each channel
        hipfftComplex* dev_freq;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)*channelCount));
        hipfftComplex* dev_freq_L = dev_freq;
        hipfftComplex* dev_freq_R = dev_freq + dataCountPerChannel+1;
        hipfftHandle plan;
        checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel*2, HIPFFT_R2C, 1));
        checkCudaErrors(hipfftExecR2C(plan, dev_real_L, dev_freq_L));
        checkCudaErrors(hipfftExecR2C(plan, dev_real_R, dev_freq_R));
        checkCudaErrors(hipfftDestroy(plan));

        // copy transformed result from device to host
        checkCudaErrors(hipMemcpy(host_freq, dev_freq, sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)*channelCount, hipMemcpyDeviceToHost));

        // free device(gpu) memory
        checkCudaErrors(hipFree(dev_pcm_scatter));
        checkCudaErrors(hipFree(dev_pcm_separate));
        checkCudaErrors(hipFree(dev_real));
        checkCudaErrors(hipFree(dev_hannFilter));
        checkCudaErrors(hipFree(dev_freq));
    }
}
void myCUDA::beforePlay_8(const hipfftComplex* host_freq, void*& host_freq_filtered, void*& host_pcm_filtered_1, void*& host_pcm_filtered_2, const float* host_freq_filter, const size_t dataCountPerChannel, const size_t channelCount)
{
    // [beforePlay]
    // (host freq)
    // -> freq filter each channel
    // (host freq filtered)
    // -> ifft each channel -> real to pcm -> 채널 믹스 -> batch 두개 분리 -> 호스트 db 에 각각 더해주기
    // (host pcm filtered)
    
    // host memory allocation
    if (host_freq == nullptr) return;
    if (host_freq_filter == nullptr) return;
    // if (host_freq_filtered == nullptr) host_freq_filtered = malloc(sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)*channelCount);
    if (host_freq_filtered == nullptr) checkCudaErrors(hipHostMalloc(&host_freq_filtered, sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)*channelCount));
    if (host_pcm_filtered_1 == nullptr) host_pcm_filtered_1 = calloc(dataCountPerChannel*channelCount, sizeof(int8_t));
    if (host_pcm_filtered_2 == nullptr) host_pcm_filtered_2 = malloc(sizeof(int8_t)*dataCountPerChannel*channelCount);

    if (channelCount == 1)
    {
        constexpr size_t threadsPerBlock = 32;

        // allocate device memory
        hipfftComplex* dev_freq;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*(dataCountPerChannel+threadsPerBlock)*channelCount));
        checkCudaErrors(hipMemcpy(dev_freq, host_freq, sizeof(hipfftComplex)*(dataCountPerChannel+1)*channelCount, hipMemcpyHostToDevice));
        float* dev_freq_filter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq_filter), sizeof(float)*(dataCountPerChannel+threadsPerBlock)*channelCount));
        checkCudaErrors(hipMemcpy(dev_freq_filter, host_freq_filter, sizeof(float)*(dataCountPerChannel+1), hipMemcpyHostToDevice));

        // freq filter
        size_t blockCount = (dataCountPerChannel+threadsPerBlock)*channelCount/threadsPerBlock;
        cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_freq, dev_freq_filter);
        hipfftComplex* dev_freq_filtered = dev_freq;

        // copy filtered freq from dev to host
        checkCudaErrors(hipMemcpy(host_freq_filtered, dev_freq_filtered, sizeof(hipfftComplex)*(dataCountPerChannel+1)*channelCount, hipMemcpyDeviceToHost));

        // ifft
        hipfftReal* dev_real_filtered;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_real_filtered), sizeof(hipfftReal)*(dataCountPerChannel*2)*channelCount));
        hipfftHandle plan;
        checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel*2, HIPFFT_C2R, 1));
        checkCudaErrors(hipfftExecC2R(plan, dev_freq_filtered, dev_real_filtered));
        checkCudaErrors(hipfftDestroy(plan));

        // real to PCM
        int8_t* dev_pcm_filtered;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_filtered), sizeof(int8_t)*(dataCountPerChannel*2)*channelCount));
        blockCount = (dataCountPerChannel*2)*channelCount/threadsPerBlock;
        cuCopyR2PCM<<<blockCount, threadsPerBlock>>>(dev_real_filtered, dev_pcm_filtered);
        
        // add each batch to host 
        int8_t* dev_pcm_filtered_new_1 = dev_pcm_filtered;
        int8_t* dev_pcm_filtered_new_2 = dev_pcm_filtered + dataCountPerChannel*channelCount;
        int8_t* dev_pcm_filtered_old_1;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_filtered_old_1), sizeof(int8_t)*dataCountPerChannel*channelCount));
        checkCudaErrors(hipMemcpy(dev_pcm_filtered_old_1, host_pcm_filtered_1, sizeof(int8_t)*(dataCountPerChannel*channelCount), hipMemcpyHostToDevice));
        blockCount = (dataCountPerChannel)*channelCount/threadsPerBlock;
        cuAddInplace<<<blockCount, threadsPerBlock>>>(dev_pcm_filtered_new_1, dev_pcm_filtered_old_1);
        int8_t* dev_pcm_filtered_sum_1 = dev_pcm_filtered_old_1;
        int8_t* dev_pcm_filtered_sum_2 = dev_pcm_filtered_new_2;
        checkCudaErrors(hipMemcpy(host_pcm_filtered_1, dev_pcm_filtered_sum_1, sizeof(int8_t)*(dataCountPerChannel*channelCount), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(host_pcm_filtered_2, dev_pcm_filtered_sum_2, sizeof(int8_t)*(dataCountPerChannel*channelCount), hipMemcpyDeviceToHost));

        // free device(gpu) memory
        checkCudaErrors(hipFree(dev_freq));
        checkCudaErrors(hipFree(dev_freq_filter));
        checkCudaErrors(hipFree(dev_real_filtered));
        checkCudaErrors(hipFree(dev_pcm_filtered));
        checkCudaErrors(hipFree(dev_pcm_filtered_old_1));
    }
    else if (channelCount == 2)
    {
        constexpr size_t threadsPerBlock = 32;

        // allocate device memory
        hipfftComplex* dev_freq;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*(dataCountPerChannel+threadsPerBlock)*channelCount));
        checkCudaErrors(hipMemcpy(dev_freq, host_freq, sizeof(hipfftComplex)*(dataCountPerChannel+1)*channelCount, hipMemcpyHostToDevice));
        float* dev_freq_filter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq_filter), sizeof(float)*(dataCountPerChannel+threadsPerBlock)*channelCount));
        float* dev_freq_filter_L = dev_freq_filter;
        float* dev_freq_filter_R = dev_freq_filter + dataCountPerChannel+1;
        checkCudaErrors(hipMemcpy(dev_freq_filter_L, host_freq_filter, sizeof(float)*(dataCountPerChannel+1), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dev_freq_filter_R, host_freq_filter, sizeof(float)*(dataCountPerChannel+1), hipMemcpyHostToDevice));

        // freq filter each channel
        size_t blockCount = (dataCountPerChannel+threadsPerBlock)*channelCount/threadsPerBlock;
        cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_freq, dev_freq_filter);
        hipfftComplex* dev_freq_filtered = dev_freq;
        hipfftComplex* dev_freq_filtered_L = dev_freq;
        hipfftComplex* dev_freq_filtered_R = dev_freq + dataCountPerChannel+1;

        // copy filtered freq from dev to host
        checkCudaErrors(hipMemcpy(host_freq_filtered, dev_freq_filtered, sizeof(hipfftComplex)*(dataCountPerChannel+1)*channelCount, hipMemcpyDeviceToHost));

        // ifft each channel
        hipfftReal* dev_real_filtered;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_real_filtered), sizeof(hipfftReal)*(dataCountPerChannel*2)*channelCount));
        hipfftReal* dev_real_filtered_L = dev_real_filtered;
        hipfftReal* dev_real_filtered_R = dev_real_filtered + dataCountPerChannel*2;
        hipfftHandle plan;
        checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel*2, HIPFFT_C2R, 1));
        checkCudaErrors(hipfftExecC2R(plan, dev_freq_filtered_L, dev_real_filtered_L));
        checkCudaErrors(hipfftExecC2R(plan, dev_freq_filtered_R, dev_real_filtered_R));
        checkCudaErrors(hipfftDestroy(plan));

        // real to PCM
        int8_t* dev_pcm_filtered;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_filtered), sizeof(int8_t)*(dataCountPerChannel*2)*channelCount));
        blockCount = (dataCountPerChannel*2)*channelCount/threadsPerBlock;
        cuCopyR2PCM<<<blockCount, threadsPerBlock>>>(dev_real_filtered, dev_pcm_filtered);
        int8_t* dev_pcm_filtered_separate = dev_pcm_filtered;

        // scatter channel LRLRLR... : 
        int8_t* dev_pcm_filtered_scatter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_filtered_scatter), sizeof(int8_t)*(dataCountPerChannel*2)*channelCount));
        cuScatterLR<<<blockCount, threadsPerBlock>>>(dev_pcm_filtered_separate, dev_pcm_filtered_scatter, dataCountPerChannel*2*channelCount);

        // add each batch to host 
        int8_t* dev_pcm_filtered_new_1 = dev_pcm_filtered_scatter;
        int8_t* dev_pcm_filtered_new_2 = dev_pcm_filtered_scatter + dataCountPerChannel*channelCount;
        int8_t* dev_pcm_filtered_old_1;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_filtered_old_1), sizeof(int8_t)*dataCountPerChannel*channelCount));
        checkCudaErrors(hipMemcpy(dev_pcm_filtered_old_1, host_pcm_filtered_1, sizeof(int8_t)*(dataCountPerChannel*channelCount), hipMemcpyHostToDevice));
        blockCount = (dataCountPerChannel)*channelCount/threadsPerBlock;
        cuAddInplace<<<blockCount, threadsPerBlock>>>(dev_pcm_filtered_new_1, dev_pcm_filtered_old_1);
        int8_t* dev_pcm_filtered_sum_1 = dev_pcm_filtered_old_1;
        int8_t* dev_pcm_filtered_sum_2 = dev_pcm_filtered_new_2;
        checkCudaErrors(hipMemcpy(host_pcm_filtered_1, dev_pcm_filtered_sum_1, sizeof(int8_t)*(dataCountPerChannel*channelCount), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(host_pcm_filtered_2, dev_pcm_filtered_sum_2, sizeof(int8_t)*(dataCountPerChannel*channelCount), hipMemcpyDeviceToHost));

        // free device(gpu) memory
        checkCudaErrors(hipFree(dev_freq));
        checkCudaErrors(hipFree(dev_freq_filter));
        checkCudaErrors(hipFree(dev_real_filtered));
        checkCudaErrors(hipFree(dev_pcm_filtered));
        checkCudaErrors(hipFree(dev_pcm_filtered_scatter));
        checkCudaErrors(hipFree(dev_pcm_filtered_old_1));
    }
}

void myCUDA::afterRecord_16(const int16_t* host_pcm1, const int16_t* host_pcm2, const float* host_hannFilter, void*& host_freq, const size_t dataCountPerChannel, const size_t channelCount)
{
    // [afterRecord]
    // (host pcm) 
    // 두개 연결 -> 채널 분리 -> 채널별 한필터 -> fft 
    // (host freq)

    // host memory allocation
    if (host_pcm1 == nullptr) return;
    if (host_pcm2 == nullptr) return;
    if (host_freq == nullptr) host_freq = malloc(sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)*channelCount);

    if (channelCount == 1)
    {
        // allocate device memory
        int16_t* dev_pcm;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm), sizeof(int16_t)*dataCountPerChannel*2));
        checkCudaErrors(hipMemcpy(dev_pcm, host_pcm1, sizeof(int16_t)*dataCountPerChannel, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dev_pcm + dataCountPerChannel, host_pcm2, sizeof(int16_t)*dataCountPerChannel, hipMemcpyHostToDevice));
        
        constexpr size_t threadsPerBlock = 32;
        size_t blockCount = dataCountPerChannel*2/threadsPerBlock;

        // PCM to Real
        hipfftReal* dev_real;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_real), sizeof(hipfftReal)*dataCountPerChannel*2));
        cuCopyPCM2R<<<blockCount, threadsPerBlock>>>(dev_pcm, dev_real, dataCountPerChannel*2);

        // Hann filter
        float* dev_hannFilter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_hannFilter), sizeof(float)*dataCountPerChannel*2));
        checkCudaErrors(hipMemcpy(dev_hannFilter, host_hannFilter, sizeof(float)*dataCountPerChannel*2, hipMemcpyHostToDevice));
        cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_real, dev_hannFilter);

        // FFT
        hipfftComplex* dev_freq;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)));
        hipfftHandle plan;
        checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel*2, HIPFFT_R2C, 1));
        checkCudaErrors(hipfftExecR2C(plan, dev_real, dev_freq));
        checkCudaErrors(hipfftDestroy(plan));

        // copy transformed result from device to host
        checkCudaErrors(hipMemcpy(host_freq, dev_freq, sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1), hipMemcpyDeviceToHost));

        // free device(gpu) memory
        checkCudaErrors(hipFree(dev_pcm));
        checkCudaErrors(hipFree(dev_real));
        checkCudaErrors(hipFree(dev_hannFilter));
        checkCudaErrors(hipFree(dev_freq));
    }
    if (channelCount == 2)
    {
        // allocate device memory
        int16_t* dev_pcm_scatter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_scatter), sizeof(int16_t)*dataCountPerChannel*channelCount*2));
        checkCudaErrors(hipMemcpy(dev_pcm_scatter, host_pcm1, sizeof(int16_t)*dataCountPerChannel*channelCount, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dev_pcm_scatter + dataCountPerChannel*channelCount, host_pcm2, sizeof(int16_t)*dataCountPerChannel*channelCount, hipMemcpyHostToDevice));
        int16_t* dev_pcm_separate;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_separate), sizeof(int16_t)*dataCountPerChannel*channelCount*2));

        // separate channel LLLLL...RRRR...
        constexpr size_t threadsPerBlock = 32;
        size_t blockCount = dataCountPerChannel*channelCount*2/threadsPerBlock;
        cuSeparateLR<<<blockCount, threadsPerBlock>>>(dev_pcm_scatter, dev_pcm_separate, dataCountPerChannel*channelCount*2);
        // int16_t* dev_pcm_L = dev_pcm_separate;
        // int16_t* dev_pcm_R = dev_pcm_separate + dataCountPerChannel*2;

        // PCM to Real
        hipfftReal* dev_real;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_real), sizeof(hipfftReal)*dataCountPerChannel*channelCount*2));
        cuCopyPCM2R<<<blockCount, threadsPerBlock>>>(dev_pcm_separate, dev_real, dataCountPerChannel*2);
        hipfftReal* dev_real_L = dev_real;
        hipfftReal* dev_real_R = dev_real + dataCountPerChannel*2;

        // Hann filter each channel
        blockCount = dataCountPerChannel*2/threadsPerBlock;
        float* dev_hannFilter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_hannFilter), sizeof(float)*dataCountPerChannel*2));
        checkCudaErrors(hipMemcpy(dev_hannFilter, host_hannFilter, sizeof(float)*dataCountPerChannel*2, hipMemcpyHostToDevice));
        cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_real_L, dev_hannFilter);
        cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_real_R, dev_hannFilter);

        // FFT each channel
        hipfftComplex* dev_freq;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)*channelCount));
        hipfftComplex* dev_freq_L = dev_freq;
        hipfftComplex* dev_freq_R = dev_freq + dataCountPerChannel+1;
        hipfftHandle plan;
        checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel*2, HIPFFT_R2C, 1));
        checkCudaErrors(hipfftExecR2C(plan, dev_real_L, dev_freq_L));
        checkCudaErrors(hipfftExecR2C(plan, dev_real_R, dev_freq_R));
        checkCudaErrors(hipfftDestroy(plan));

        // copy transformed result from device to host
        checkCudaErrors(hipMemcpy(host_freq, dev_freq, sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)*channelCount, hipMemcpyDeviceToHost));

        // free device(gpu) memory
        checkCudaErrors(hipFree(dev_pcm_scatter));
        checkCudaErrors(hipFree(dev_pcm_separate));
        checkCudaErrors(hipFree(dev_real));
        checkCudaErrors(hipFree(dev_hannFilter));
        checkCudaErrors(hipFree(dev_freq));
    }
}
void myCUDA::beforePlay_16(const hipfftComplex* host_freq, void*& host_freq_filtered, void*& host_pcm_filtered_1, void*& host_pcm_filtered_2, const float* host_freq_filter, const size_t dataCountPerChannel, const size_t channelCount)
{
    // [beforePlay]
    // (host freq)
    // -> freq filter each channel
    // (host freq filtered)
    // -> ifft each channel -> real to pcm -> 채널 믹스 -> batch 두개 분리 -> 호스트 db 에 각각 더해주기
    // (host pcm filtered)
    
    // host memory allocation
    if (host_freq == nullptr) return;
    if (host_freq_filter == nullptr) return;
    if (host_freq_filtered == nullptr) host_freq_filtered = malloc(sizeof(hipfftComplex)*(dataCountPerChannel*2/2+1)*channelCount);
    if (host_pcm_filtered_1 == nullptr) host_pcm_filtered_1 = calloc(dataCountPerChannel*channelCount, sizeof(int16_t));
    if (host_pcm_filtered_2 == nullptr) host_pcm_filtered_2 = malloc(sizeof(int16_t)*dataCountPerChannel*channelCount);


    if (channelCount == 1)
    {
        constexpr size_t threadsPerBlock = 32;

        // allocate device memory
        hipfftComplex* dev_freq;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*(dataCountPerChannel+threadsPerBlock)*channelCount));
        checkCudaErrors(hipMemcpy(dev_freq, host_freq, sizeof(hipfftComplex)*(dataCountPerChannel+1)*channelCount, hipMemcpyHostToDevice));
        float* dev_freq_filter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq_filter), sizeof(float)*(dataCountPerChannel+threadsPerBlock)*channelCount));
        checkCudaErrors(hipMemcpy(dev_freq_filter, host_freq_filter, sizeof(float)*(dataCountPerChannel+1), hipMemcpyHostToDevice));

        // freq filter
        size_t blockCount = (dataCountPerChannel+threadsPerBlock)*channelCount/threadsPerBlock;
        cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_freq, dev_freq_filter);
        hipfftComplex* dev_freq_filtered = dev_freq;

        // copy filtered freq from dev to host
        checkCudaErrors(hipMemcpy(host_freq_filtered, dev_freq_filtered, sizeof(hipfftComplex)*(dataCountPerChannel+1)*channelCount, hipMemcpyDeviceToHost));

        // ifft
        hipfftReal* dev_real_filtered;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_real_filtered), sizeof(hipfftReal)*(dataCountPerChannel*2)*channelCount));
        hipfftHandle plan;
        checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel*2, HIPFFT_C2R, 1));
        checkCudaErrors(hipfftExecC2R(plan, dev_freq_filtered, dev_real_filtered));
        checkCudaErrors(hipfftDestroy(plan));

        // real to PCM
        int16_t* dev_pcm_filtered;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_filtered), sizeof(int16_t)*(dataCountPerChannel*2)*channelCount));
        blockCount = (dataCountPerChannel*2)*channelCount/threadsPerBlock;
        cuCopyR2PCM<<<blockCount, threadsPerBlock>>>(dev_real_filtered, dev_pcm_filtered);

        // add each batch to host 
        int16_t* dev_pcm_filtered_new_1 = dev_pcm_filtered;
        int16_t* dev_pcm_filtered_new_2 = dev_pcm_filtered + dataCountPerChannel*channelCount;
        int16_t* dev_pcm_filtered_old_1;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_filtered_old_1), sizeof(int16_t)*dataCountPerChannel*channelCount));
        checkCudaErrors(hipMemcpy(dev_pcm_filtered_old_1, host_pcm_filtered_1, sizeof(int16_t)*(dataCountPerChannel*channelCount), hipMemcpyHostToDevice));
        blockCount = (dataCountPerChannel)*channelCount/threadsPerBlock;
        cuAddInplace<<<blockCount, threadsPerBlock>>>(dev_pcm_filtered_new_1, dev_pcm_filtered_old_1);
        int16_t* dev_pcm_filtered_sum_1 = dev_pcm_filtered_old_1;
        int16_t* dev_pcm_filtered_sum_2 = dev_pcm_filtered_new_2;
        checkCudaErrors(hipMemcpy(host_pcm_filtered_1, dev_pcm_filtered_sum_1, sizeof(int16_t)*(dataCountPerChannel*channelCount), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(host_pcm_filtered_2, dev_pcm_filtered_sum_2, sizeof(int16_t)*(dataCountPerChannel*channelCount), hipMemcpyDeviceToHost));

        // free device(gpu) memory
        checkCudaErrors(hipFree(dev_freq));
        checkCudaErrors(hipFree(dev_freq_filter));
        checkCudaErrors(hipFree(dev_real_filtered));
        checkCudaErrors(hipFree(dev_pcm_filtered));
        checkCudaErrors(hipFree(dev_pcm_filtered_old_1));
    }
    else if (channelCount == 2)
    {
        constexpr size_t threadsPerBlock = 32;

        // allocate device memory
        hipfftComplex* dev_freq;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*(dataCountPerChannel+threadsPerBlock)*channelCount));
        checkCudaErrors(hipMemcpy(dev_freq, host_freq, sizeof(hipfftComplex)*(dataCountPerChannel+1)*channelCount, hipMemcpyHostToDevice));
        float* dev_freq_filter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq_filter), sizeof(float)*(dataCountPerChannel+threadsPerBlock)*channelCount));
        float* dev_freq_filter_L = dev_freq_filter;
        float* dev_freq_filter_R = dev_freq_filter + dataCountPerChannel+1;
        checkCudaErrors(hipMemcpy(dev_freq_filter_L, host_freq_filter, sizeof(float)*(dataCountPerChannel+1), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(dev_freq_filter_R, host_freq_filter, sizeof(float)*(dataCountPerChannel+1), hipMemcpyHostToDevice));

        // freq filter each channel
        size_t blockCount = (dataCountPerChannel+threadsPerBlock)*channelCount/threadsPerBlock;
        cuMultFilterInplace<<<blockCount, threadsPerBlock>>>(dev_freq, dev_freq_filter);
        hipfftComplex* dev_freq_filtered = dev_freq;
        hipfftComplex* dev_freq_filtered_L = dev_freq;
        hipfftComplex* dev_freq_filtered_R = dev_freq + dataCountPerChannel+1;

        // copy filtered freq from dev to host
        checkCudaErrors(hipMemcpy(host_freq_filtered, dev_freq_filtered, sizeof(hipfftComplex)*(dataCountPerChannel+1)*channelCount, hipMemcpyDeviceToHost));

        // ifft each channel
        hipfftReal* dev_real_filtered;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_real_filtered), sizeof(hipfftReal)*(dataCountPerChannel*2)*channelCount));
        hipfftReal* dev_real_filtered_L = dev_real_filtered;
        hipfftReal* dev_real_filtered_R = dev_real_filtered + dataCountPerChannel*2;
        hipfftHandle plan;
        checkCudaErrors(hipfftPlan1d(&plan, dataCountPerChannel*2, HIPFFT_C2R, 1));
        checkCudaErrors(hipfftExecC2R(plan, dev_freq_filtered_L, dev_real_filtered_L));
        checkCudaErrors(hipfftExecC2R(plan, dev_freq_filtered_R, dev_real_filtered_R));
        checkCudaErrors(hipfftDestroy(plan));

        // real to PCM
        int16_t* dev_pcm_filtered;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_filtered), sizeof(int16_t)*(dataCountPerChannel*2)*channelCount));
        blockCount = (dataCountPerChannel*2)*channelCount/threadsPerBlock;
        cuCopyR2PCM<<<blockCount, threadsPerBlock>>>(dev_real_filtered, dev_pcm_filtered);
        int16_t* dev_pcm_filtered_separate = dev_pcm_filtered;

        // scatter channel LRLRLR... : 
        int16_t* dev_pcm_filtered_scatter;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_filtered_scatter), sizeof(int16_t)*(dataCountPerChannel*2)*channelCount));
        cuScatterLR<<<blockCount, threadsPerBlock>>>(dev_pcm_filtered_separate, dev_pcm_filtered_scatter, dataCountPerChannel*2*channelCount);

        // add each batch to host 
        int16_t* dev_pcm_filtered_new_1 = dev_pcm_filtered_scatter;
        int16_t* dev_pcm_filtered_new_2 = dev_pcm_filtered_scatter + dataCountPerChannel*channelCount;
        int16_t* dev_pcm_filtered_old_1;
        checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm_filtered_old_1), sizeof(int16_t)*dataCountPerChannel*channelCount));
        checkCudaErrors(hipMemcpy(dev_pcm_filtered_old_1, host_pcm_filtered_1, sizeof(int16_t)*(dataCountPerChannel*channelCount), hipMemcpyHostToDevice));
        blockCount = (dataCountPerChannel)*channelCount/threadsPerBlock;
        cuAddInplace<<<blockCount, threadsPerBlock>>>(dev_pcm_filtered_new_1, dev_pcm_filtered_old_1);
        int16_t* dev_pcm_filtered_sum_1 = dev_pcm_filtered_old_1;
        int16_t* dev_pcm_filtered_sum_2 = dev_pcm_filtered_new_2;
        checkCudaErrors(hipMemcpy(host_pcm_filtered_1, dev_pcm_filtered_sum_1, sizeof(int16_t)*(dataCountPerChannel*channelCount), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(host_pcm_filtered_2, dev_pcm_filtered_sum_2, sizeof(int16_t)*(dataCountPerChannel*channelCount), hipMemcpyDeviceToHost));

        // free device(gpu) memory
        checkCudaErrors(hipFree(dev_freq));
        checkCudaErrors(hipFree(dev_freq_filter));
        checkCudaErrors(hipFree(dev_real_filtered));
        checkCudaErrors(hipFree(dev_pcm_filtered));
        checkCudaErrors(hipFree(dev_pcm_filtered_scatter));
        checkCudaErrors(hipFree(dev_pcm_filtered_old_1));
    }
}

void myCUDA::freePtr(void*& ptr)
{
    if (ptr == nullptr) return;
    // free(ptr);
    hipHostFree(ptr);
    ptr = nullptr;
}

void myCUDA::logSqrtComplexBatch_8(const hipfftComplex* host_freq, float* host_logSqrt_freq, const size_t dataCountPerChannel)
{
    if (host_freq == nullptr) return;
    if (host_logSqrt_freq == nullptr) return;
    
    // device memory allocation
    hipfftComplex* dev_freq;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*dataCountPerChannel));
    checkCudaErrors(hipMemcpy(dev_freq, host_freq+1, sizeof(hipfftComplex)*dataCountPerChannel, hipMemcpyHostToDevice));
    float* dev_logSqrt_freq;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_logSqrt_freq), sizeof(float)*dataCountPerChannel));

    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = dataCountPerChannel/threadsPerBlock;

    // log2f(hypotf(complex))/7
    cuSqrtComplex_8<<<blockCount, threadsPerBlock>>>(dev_freq, dev_logSqrt_freq);
    
    // copy transformed result from device to host
    checkCudaErrors(hipMemcpy(host_logSqrt_freq, dev_logSqrt_freq, sizeof(float)*(dataCountPerChannel), hipMemcpyDeviceToHost));

    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_freq));
    checkCudaErrors(hipFree(dev_logSqrt_freq));
}
void myCUDA::logSqrtComplexBatch_16(const hipfftComplex* host_freq, float* host_logSqrt_freq, const size_t dataCountPerChannel)
{
    if (host_freq == nullptr) return;
    if (host_logSqrt_freq == nullptr) return;
    
    // device memory allocation
    hipfftComplex* dev_freq;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*dataCountPerChannel));
    checkCudaErrors(hipMemcpy(dev_freq, host_freq+1, sizeof(hipfftComplex)*dataCountPerChannel, hipMemcpyHostToDevice));
    float* dev_logSqrt_freq;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_logSqrt_freq), sizeof(float)*dataCountPerChannel));

    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = dataCountPerChannel/threadsPerBlock;

    // log2f(hypotf(complex))/15
    cuSqrtComplex_16<<<blockCount, threadsPerBlock>>>(dev_freq, dev_logSqrt_freq);
    
    // copy transformed result from device to host
    checkCudaErrors(hipMemcpy(host_logSqrt_freq, dev_logSqrt_freq, sizeof(float)*(dataCountPerChannel), hipMemcpyDeviceToHost));

    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_freq));
    checkCudaErrors(hipFree(dev_logSqrt_freq));
}

void myCUDA::initFreqFilter(float* host_freqFilter, const float initValue, const size_t filterSize)
{
    if (host_freqFilter == nullptr) return;

    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = filterSize/threadsPerBlock + 1;
    
    // device memory allocation
    float* dev_freqFilter;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freqFilter), sizeof(float)*blockCount*threadsPerBlock));
    checkCudaErrors(hipMemcpy(dev_freqFilter, host_freqFilter, sizeof(float)*filterSize, hipMemcpyHostToDevice));
    
    // cuMultFloat
    cuInitFloatFilter<<<blockCount, threadsPerBlock>>>(dev_freqFilter, initValue);

    // memcpy device to host
    checkCudaErrors(hipMemcpy(host_freqFilter, dev_freqFilter, sizeof(float)*filterSize, hipMemcpyDeviceToHost));
    
    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_freqFilter));
}
void myCUDA::setFreqFilter(float* host_freqFilter, const float weight, const size_t freqIdxSt, const size_t freqIdxEn)
{
    if (host_freqFilter == nullptr) return;

    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = (freqIdxEn - freqIdxSt + 1)/threadsPerBlock + 1;

    // device memory allocation
    float* dev_freqFilter;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freqFilter), sizeof(float)*blockCount*threadsPerBlock));
    checkCudaErrors(hipMemcpy(dev_freqFilter, host_freqFilter+freqIdxSt, sizeof(float)*(freqIdxEn - freqIdxSt + 1), hipMemcpyHostToDevice));
    
    // cuMultFloat
    cuMultFloatWeight<<<blockCount, threadsPerBlock>>>(dev_freqFilter, weight);

    // memcpy device to host
    checkCudaErrors(hipMemcpy(host_freqFilter+freqIdxSt, dev_freqFilter, sizeof(float)*(freqIdxEn - freqIdxSt + 1), hipMemcpyDeviceToHost));
    
    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_freqFilter));
}

void myCUDA::normPCMBatch_8(const uint8_t* host_pcm, float* host_norm_pcm, const size_t dataCountPerChannel)
{
    auto mallocSt = std::chrono::steady_clock::now();
    if (host_pcm == nullptr) return;
    if (host_norm_pcm == nullptr) return;
    
    // device memory allocation
    uint8_t* dev_pcm;
    hipMalloc(reinterpret_cast<void **>(&dev_pcm), sizeof(uint8_t)*dataCountPerChannel);
    hipMemcpy(dev_pcm, host_pcm, sizeof(uint8_t)*dataCountPerChannel, hipMemcpyHostToDevice);
    float* dev_norm_pcm;
    hipMalloc(reinterpret_cast<void **>(&dev_norm_pcm), sizeof(float)*dataCountPerChannel);

    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = dataCountPerChannel/threadsPerBlock;

    // pcm/2^8
    cuNormPCM_8<<<blockCount, threadsPerBlock>>>(dev_pcm, dev_norm_pcm);
    
    // copy transformed result from device to host
    hipMemcpy(host_norm_pcm, dev_norm_pcm, sizeof(float)*(dataCountPerChannel), hipMemcpyDeviceToHost);

    // free device(gpu) memory
    hipFree(dev_pcm);
    hipFree(dev_norm_pcm);
    auto mallocEn = std::chrono::steady_clock::now();
    auto mallocDuration = mallocEn-mallocSt;
    std::cout << "\tcuda total duration [ns] : " << mallocDuration.count() << std::endl;
}
void myCUDA::normPCMBatch_16(const int16_t* host_pcm, float* host_norm_pcm, const size_t dataCountPerChannel)
{
    if (host_pcm == nullptr) return;
    if (host_norm_pcm == nullptr) return;
    
    // device memory allocation
    int16_t* dev_pcm;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_pcm), sizeof(int16_t)*dataCountPerChannel));
    checkCudaErrors(hipMemcpy(dev_pcm, host_pcm, sizeof(int16_t)*dataCountPerChannel, hipMemcpyHostToDevice));
    float* dev_norm_pcm;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_norm_pcm), sizeof(float)*dataCountPerChannel));

    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = dataCountPerChannel/threadsPerBlock;

    // pcm/2^15
    cuNormPCM_16<<<blockCount, threadsPerBlock>>>(dev_pcm, dev_norm_pcm);
    
    // copy transformed result from device to host
    checkCudaErrors(hipMemcpy(host_norm_pcm, dev_norm_pcm, sizeof(float)*(dataCountPerChannel), hipMemcpyDeviceToHost));

    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_pcm));
    checkCudaErrors(hipFree(dev_norm_pcm));
}

void myCUDA::normSqrtComplexBatch_8(const hipfftComplex* host_freq, float* host_normSqrt_freq, const size_t dataCountPerChannel)
{
    if (host_freq == nullptr) return;
    if (host_normSqrt_freq == nullptr) return;
    
    // device memory allocation
    hipfftComplex* dev_freq;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*dataCountPerChannel));
    checkCudaErrors(hipMemcpy(dev_freq, host_freq+1, sizeof(hipfftComplex)*dataCountPerChannel, hipMemcpyHostToDevice));
    float* dev_normSqrt_freq;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_normSqrt_freq), sizeof(float)*dataCountPerChannel));

    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = dataCountPerChannel/threadsPerBlock;

    // pcm/2^7
    cuSqrtComplex_8<<<blockCount, threadsPerBlock>>>(dev_freq, dev_normSqrt_freq);
    
    // copy transformed result from device to host
    checkCudaErrors(hipMemcpy(host_normSqrt_freq, dev_normSqrt_freq, sizeof(float)*(dataCountPerChannel), hipMemcpyDeviceToHost));

    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_freq));
    checkCudaErrors(hipFree(dev_normSqrt_freq));
}
void myCUDA::normSqrtComplexBatch_16(const hipfftComplex* host_freq, float* host_normSqrt_freq, const size_t dataCountPerChannel)
{
    if (host_freq == nullptr) return;
    if (host_normSqrt_freq == nullptr) return;
    
    // device memory allocation
    hipfftComplex* dev_freq;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_freq), sizeof(hipfftComplex)*dataCountPerChannel));
    checkCudaErrors(hipMemcpy(dev_freq, host_freq+1, sizeof(hipfftComplex)*dataCountPerChannel, hipMemcpyHostToDevice));
    float* dev_normSqrt_freq;
    checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&dev_normSqrt_freq), sizeof(float)*dataCountPerChannel));

    constexpr size_t threadsPerBlock = 32;
    size_t blockCount = dataCountPerChannel/threadsPerBlock;

    // pcm/2^7
    cuSqrtComplex_16<<<blockCount, threadsPerBlock>>>(dev_freq, dev_normSqrt_freq);
    
    // copy transformed result from device to host
    checkCudaErrors(hipMemcpy(host_normSqrt_freq, dev_normSqrt_freq, sizeof(float)*(dataCountPerChannel), hipMemcpyDeviceToHost));

    // free device(gpu) memory
    checkCudaErrors(hipFree(dev_freq));
    checkCudaErrors(hipFree(dev_normSqrt_freq));
}

// todo : 가독성 개선 측면에서 void* 다시 원래대로 복구 후 테스트해보기
// todo : frequency complex 에서 frequency sqrt value glfloat 로 변환하는 __global__ 만들기, 이를 사용하는 __host__ 만들기, 성능테스트로 비교해보기 : 250,000
